
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void foo()
{
    if (blockIdx.x == 0 && threadIdx.x == 0)
    {
        printf("Kernel running. All good :)\n");
    }
}

int main()
{
    // <<<numBlocks, threadsPerBlock, sizeOfSharedMemory, cudaStream>>>
    foo<<<1, 1>>>();
    hipDeviceSynchronize();
    return 0;
}